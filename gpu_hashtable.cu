#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

// Thomas Mueller in https://stackoverflow.com/questions/664014/what-integer-hash-function-are-good-that-accepts-an-integer-hash-key
__device__ int getHash(int key, int capacity) {
	key = ((key >> 16) ^ key) * 0x45d9f3b;
    key = ((key >> 16) ^ key) * 0x45d9f3b;
    key = (key >> 16) ^ key;

	return key % capacity;
}

__global__ void kernel_insert_key(int *keys, int* values, int numKeys, HashTable* hashTable) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= numKeys) {
		return;
	}

	
	// Obtin elementul din array unde trebuie adaugat elementul
	int hashcode = getHash(keys[idx], hashTable->capacity);
	bool foundEmptySlot = false;
	int old = 0;
	
	// Cauta primul slot liber din array
	while (!foundEmptySlot) {
		// Obtine atomic elementul de pe slotul incercat 
		int old = atomicCAS(&hashTable->elements[hashcode].key, EMPTY_SLOT, keys[idx]);
		if (old == EMPTY_SLOT || old == keys[idx]) {
			atomicCAS(&hashTable->elements[hashcode].value, EMPTY_SLOT, values[idx]);
			foundEmptySlot = true;
		}
		// Trece la slotul urmator daca cel curent este ocupat
		hashcode = (hashcode + 1)  % (hashTable->capacity - 1);
	}

	// Mareste size-ul doar daca elementul a fost adaugat pe un slot gol
	if (old == EMPTY_SLOT) {
		atomicAdd(&hashTable->size, 1);
	}
	
	//printf("%d %d %d %d\n", idx, hashcode, hashTable->elements[hashcode].key, hashTable->elements[hashcode].value);
		
}

__global__ void kernel_get(HashTable* hashTable, int* keys, int* values, int numKeys) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= numKeys) {
		return;
	}

	int hashcode = getHash(keys[idx], hashTable->capacity);
	bool foundKey = false;

	while(!foundKey) {
		if (hashTable->elements[hashcode].key == keys[hashcode]) {
			values[hashcode] = hashTable->elements[hashcode].value;
			foundKey = true;
		} else if (hashTable->elements[hashcode].key == EMPTY_SLOT) {
			values[hashcode] = EMPTY_SLOT;
			break;
		}
		hashcode = (hashcode + 1)  % (hashTable->capacity - 1);
	}

}

__global__ void kernel_reshape(Elem* newElements, int newCapacity, HashTable* oldHashTable) {

	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= oldHashTable->capacity - 1) {
		return;
	}

	int hashcode = getHash(oldHashTable->elements[idx].key, newCapacity);

	int oldValue = atomicCAS(&newElements[hashcode].key, EMPTY_SLOT, oldHashTable->elements[hashcode].key);
	if (oldValue == EMPTY_SLOT) {
		atomicCAS(&newElements[hashcode].value, EMPTY_SLOT, oldHashTable->elements[hashcode].value);
		//atomicAdd(&resizedHashTable->size, 1);
	}

	////printf("%d %d %d %d\n",idx, hashcode, newElements[hashcode].key, newElements[hashcode].value);

}

__global__ void kernel_test(Elem* newElements, int size) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (idx >= size) {
		return;
	}

	printf("%d %d\n", newElements[idx].key, newElements[idx].value);
}



/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	hipError_t rc;
	
	rc = glbGpuAllocator->_cudaMallocManaged((void**) &hashTable, sizeof(HashTable));
	if (rc != hipSuccess) {
		cout << "HashMap Malloc Failed!" << endl;
		return;
	}

	// Numarul maxim de elemente din hashtable
	hashTable->capacity = size;
	// // Numarul de elemente ocupate din hashtable
	hashTable->size = 0;

	//Aloc array-ul de bucket-uri (array de liste)
	rc = glbGpuAllocator->_cudaMalloc((void**) &(hashTable->elements), size * sizeof(Elem));
	if (rc != hipSuccess) {
		cout << "Elements Malloc Failed!" << endl;
		return;
	}


}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(hashTable->elements);
	glbGpuAllocator->_cudaFree(hashTable);
}


/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	hipError_t rc;
	int numBlocks;

	Elem* newElements;
	rc = glbGpuAllocator->_cudaMalloc((void**) &newElements, numBucketsReshape * sizeof(Elem));
	if (rc != hipSuccess) {
		cout << "Elements Malloc Failed!" << endl;
		return;
	}
	

	if (hashTable->size == 0) {
		hashTable->elements = newElements;
		hashTable->capacity = numBucketsReshape;
		//cout << "era gol" << endl;
		return;
	}

 	numBlocks = hashTable->size / BLOCK_SIZE;

	// Caz in care block-ul final nu este complet
	if (hashTable->size % BLOCK_SIZE) {
		numBlocks++;
	}
	
	
	kernel_reshape<<<numBlocks, BLOCK_SIZE>>> (newElements, numBucketsReshape, hashTable);
	hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
	}
	

	 glbGpuAllocator->_cudaFree(hashTable->elements);
	 //glbGpuAllocator->_cudaFree(hashTable);
	hashTable->elements = newElements;
	hashTable->capacity = numBucketsReshape; 

}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	hipError_t rc;
	int numBlocks;
	// Aloca memorie pentru array-urile de chei si de valori in GPU
	int *keysDev, *valuesDev;

	rc = glbGpuAllocator->_cudaMalloc((void **) &keysDev, numKeys * sizeof(int));
	if (rc != hipSuccess) {
		perror("keysDev Malloc Failed!");
		return false;
	}
	rc = glbGpuAllocator->_cudaMalloc((void **) &valuesDev, numKeys * sizeof(int));
	if (rc != hipSuccess) {
		perror("valuesDev Malloc Failed!");
		return false;
	}
	
	hipMemcpy(keysDev, keys , numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(valuesDev, values, numKeys * sizeof(int), hipMemcpyHostToDevice);

	numBlocks = numKeys / BLOCK_SIZE;

	// Caz in care block-ul final nu este complet
	if (numKeys % BLOCK_SIZE) {
		numBlocks++;
	}

	// Caz cand e nevoie de rehash
	if ((float)(hashTable->size + numKeys) / hashTable->capacity >= MAX_LOAD_FACTOR) {
		// Calculeaza noua capacitate
		//int updatedCapacity = ((float)(hashTable->size + numKeys) / MAX_LOAD_FACTOR) + 1;
		//cout << "Trebuie resize "<< updatedCapacity << endl;
		//reshape(updatedCapacity);
		//cout << hashTable->size << " " << hashTable->capacity << endl;
		
	}
	
	
	kernel_insert_key<<<numBlocks, BLOCK_SIZE>>> (keysDev, valuesDev, numKeys, hashTable);
	hipDeviceSynchronize();
	//cout << hashTable->size << " " << hashTable->capacity << endl;
	

	glbGpuAllocator->_cudaFree(keysDev);
	glbGpuAllocator->_cudaFree(valuesDev);

	return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	hipError_t rc;
	int numBlocks;
	int *keysDev, *values;


	rc = glbGpuAllocator->_cudaMalloc((void **) &keysDev, numKeys * sizeof(int));
	if (rc != hipSuccess) {
		perror("keysDev Malloc Failed!");
		return NULL;
	}
	rc = glbGpuAllocator->_cudaMallocManaged((void **) &values, numKeys * sizeof(int));
	if (rc != hipSuccess) {
		perror("valuesDev Malloc Failed!");
		return NULL;
	}
	
	hipMemcpy(keysDev, keys , numKeys * sizeof(int), hipMemcpyHostToDevice);

	numBlocks = numKeys / BLOCK_SIZE;

	// Caz in care block-ul final nu este complet
	if (numKeys % BLOCK_SIZE) {
		numBlocks++;
	}
	kernel_get<<<numBlocks, BLOCK_SIZE>>> (hashTable, keysDev, values, numKeys);
	hipDeviceSynchronize();

	glbGpuAllocator->_cudaFree(keysDev);

	return values;
}
