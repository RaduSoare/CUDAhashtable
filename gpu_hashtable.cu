#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

// Thomas Mueller in https://stackoverflow.com/questions/664014/what-integer-hash-function-are-good-that-accepts-an-integer-hash-key
__device__ int getHash(int key, int capacity) {
	key = ((key >> 16) ^ key) * 0x45d9f3b;
    key = ((key >> 16) ^ key) * 0x45d9f3b;
    key = (key >> 16) ^ key;

	return key % capacity;
}

__global__ void kernel_insert_key(int *keys, int* values, int numKeys, HashTable* hashTable) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= numKeys) {
		return;
	}

	
	// Obtin elementul din array unde trebuie adaugat elementul
	int hashcode = getHash(keys[idx], hashTable->capacity);
	bool foundEmptySlot = false;
	int old = 0;
	
	// Cauta primul slot liber din array
	while (!foundEmptySlot) {
		// Obtine atomic elementul de pe slotul incercat 
		int old = atomicCAS(&hashTable->elements[hashcode].key, EMPTY_SLOT, keys[idx]);
		if (old == EMPTY_SLOT || old == keys[idx]) {
			atomicCAS(&hashTable->elements[hashcode].value, EMPTY_SLOT, values[idx]);
			foundEmptySlot = true;
		}
		// Trece la slotul urmator daca cel curent este ocupat
		hashcode = (hashcode + 1)  % (hashTable->capacity - 1);
	}

	// Mareste size-ul doar daca elementul a fost adaugat pe un slot gol
	if (old == EMPTY_SLOT) {
		atomicAdd(&hashTable->size, 1);
	}
	
	//printf("%d %d %d %d\n", idx, hashcode, hashTable->elements[hashcode].key, hashTable->elements[hashcode].value);
		
}

__global__ void kernel_get(HashTable* hashTable, int* keys, int* values, int numKeys) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= numKeys) {
		return;
	}

	int hashcode = getHash(keys[idx], hashTable->capacity);

	bool foundKey = false;
	while(!foundKey) {
		if (hashTable->elements[hashcode].key == keys[idx]) {
			values[idx] = hashTable->elements[hashcode].value;
			foundKey = true;
		}
		hashcode = (hashcode + 1)  % (hashTable->capacity - 1);
	}
	

}

__global__ void kernel_reshape(Elem* newElements, int newCapacity, HashTable* oldHashTable) {

	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= oldHashTable->capacity) {
		return;
	}
	

	int hashcode = getHash(oldHashTable->elements[idx].key, newCapacity);
	//printf("%d %d %d %d\n",idx, hashcode, oldHashTable->elements[idx].key, oldHashTable->elements[idx].value);
	bool rehashedKey = false;

	while(!rehashedKey) {
		int old = atomicCAS(&newElements[hashcode].key, EMPTY_SLOT, oldHashTable->elements[idx].key);
		if (old == EMPTY_SLOT) {
			atomicCAS(&newElements[hashcode].value, EMPTY_SLOT, oldHashTable->elements[idx].value);
			rehashedKey = true;
		}
		hashcode = (hashcode + 1)  % (newCapacity - 1);
	}

}




/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	hipError_t rc;
	
	rc = glbGpuAllocator->_cudaMallocManaged((void**) &hashTable, sizeof(HashTable));
	if (rc != hipSuccess) {
		cout << "HashMap Malloc Failed!" << endl;
		return;
	}

	// Numarul maxim de elemente din hashtable
	hashTable->capacity = size;
	// // Numarul de elemente ocupate din hashtable
	hashTable->size = 0;

	//Aloc array-ul de bucket-uri (array de liste)
	rc = glbGpuAllocator->_cudaMalloc((void**) &(hashTable->elements), size * sizeof(Elem));
	if (rc != hipSuccess) {
		cout << "Elements Malloc Failed!" << endl;
		return;
	}


}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(hashTable->elements);
	glbGpuAllocator->_cudaFree(hashTable);
}


/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	hipError_t rc;
	int numBlocks;

	Elem* newElements;
	rc = glbGpuAllocator->_cudaMalloc((void**) &newElements, numBucketsReshape * sizeof(Elem));
	if (rc != hipSuccess) {
		cout << "Elements Malloc Failed!" << endl;
		return;
	}
	

	if (hashTable->size == 0) {
		glbGpuAllocator->_cudaFree(hashTable->elements);
		hashTable->elements = newElements;
		hashTable->capacity = numBucketsReshape;
		cout << "era gol" << endl;

		return;
	}
	
 	numBlocks = hashTable->capacity / BLOCK_SIZE;
	
	// Caz in care block-ul final nu este complet
	if (hashTable->capacity % BLOCK_SIZE) {
		numBlocks++;
	}
	
	
	kernel_reshape<<<numBlocks, BLOCK_SIZE>>> (newElements, numBucketsReshape, hashTable);
	hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
	}
	

	glbGpuAllocator->_cudaFree(hashTable->elements);
	 //glbGpuAllocator->_cudaFree(hashTable);
	hashTable->elements = newElements;
	hashTable->capacity = numBucketsReshape; 

}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	hipError_t rc;
	int numBlocks;
	// Aloca memorie pentru array-urile de chei si de valori in GPU
	int *keysDev, *valuesDev;


	rc = glbGpuAllocator->_cudaMalloc((void **) &keysDev, numKeys * sizeof(int));
	if (rc != hipSuccess) {
		perror("keysDev Malloc Failed!");
		return false;
	}
	rc = glbGpuAllocator->_cudaMalloc((void **) &valuesDev, numKeys * sizeof(int));
	if (rc != hipSuccess) {
		perror("valuesDev Malloc Failed!");
		return false;
	}
	
	hipMemcpy(keysDev, keys , numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(valuesDev, values, numKeys * sizeof(int), hipMemcpyHostToDevice);

	numBlocks = numKeys / BLOCK_SIZE;

	// Caz in care block-ul final nu este complet
	if (numKeys % BLOCK_SIZE) {
		numBlocks++;
	}

	// Caz cand e nevoie de rehash
	if ((float)(hashTable->size + numKeys) / hashTable->capacity >= MAX_LOAD_FACTOR) {
		// Calculeaza noua capacitate
		int updatedCapacity = ((float)(hashTable->size + numKeys) / MAX_LOAD_FACTOR) + 1;
		cout << "Trebuie resize "<< updatedCapacity << endl;
		fprintf(stdout, "%d\n", updatedCapacity);
		reshape(updatedCapacity);
		
		
	}
	
	//cout << hashTable->size << " " << hashTable->capacity << endl;
	kernel_insert_key<<<numBlocks, BLOCK_SIZE>>> (keysDev, valuesDev, numKeys, hashTable);
	hipDeviceSynchronize();
	//cout << hashTable->size << " " << hashTable->capacity << endl;
	

	glbGpuAllocator->_cudaFree(keysDev);
	glbGpuAllocator->_cudaFree(valuesDev);

	return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	hipError_t rc;
	int numBlocks;
	int *keysDev, *values;


	rc = glbGpuAllocator->_cudaMalloc((void **) &keysDev, numKeys * sizeof(int));
	if (rc != hipSuccess) {
		perror("keysDev Malloc Failed!");
		return NULL;
	}
	rc = glbGpuAllocator->_cudaMallocManaged((void **) &values, numKeys * sizeof(int));
	if (rc != hipSuccess) {
		perror("valuesDev Malloc Failed!");
		return NULL;
	}
	
	// for (int i = 0; i < numKeys; i++) {
	// 	cout << keys[i] << endl;
	// }

	hipMemcpy(keysDev, keys , numKeys * sizeof(int), hipMemcpyHostToDevice);

	numBlocks = numKeys / BLOCK_SIZE;

	// Caz in care block-ul final nu este complet
	if (numKeys % BLOCK_SIZE) {
		numBlocks++;
	}
	kernel_get<<<numBlocks, BLOCK_SIZE>>> (hashTable, keysDev, values, numKeys);
	hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
	}

	glbGpuAllocator->_cudaFree(keysDev);

	return values;
}
